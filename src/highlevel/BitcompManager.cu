#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <memory>

#include "Check.h"
#include "CudaUtils.h"
#include "common.h"
#include "nvcomp_common_deps/hlif_shared_types.hpp"
#include "nvcomp/nvcompManager.hpp"
#include "highlevel/ManagerBase.hpp"
#include "nvcomp/bitcomp.hpp"
#include "BitcompManager.hpp"
/******************************************************************************
 * added bu Boyuan ************************************************************
 *****************************************************************************/
#include <hip/hip_fp16.h>

#ifdef ENABLE_BITCOMP

#include <bitcomp.h>

namespace nvcomp {

  // Convert the NVCOMP type to a BITCOMP type
  bitcompDataType_t bitcomp_data_type (nvcompType_t data_type)
  {
    switch (data_type) {
    case NVCOMP_TYPE_CHAR:
      return BITCOMP_SIGNED_8BIT;
      break;
    case NVCOMP_TYPE_USHORT:
      return BITCOMP_UNSIGNED_16BIT;
      break;
    case NVCOMP_TYPE_SHORT:
      return BITCOMP_SIGNED_16BIT;
      break;
    case NVCOMP_TYPE_UINT:
      return BITCOMP_UNSIGNED_32BIT;
      break;
    case NVCOMP_TYPE_INT:
      return BITCOMP_SIGNED_32BIT;
      break;
    case NVCOMP_TYPE_ULONGLONG:
      return BITCOMP_UNSIGNED_64BIT;
      break;
    case NVCOMP_TYPE_LONGLONG:
      return BITCOMP_SIGNED_64BIT;
      break;
    default:
      return BITCOMP_UNSIGNED_8BIT;
    }
  }

  /**
   * @brief Single-threaded kernel to update the common header 
   * 
   * @param common_header header filled in by this routine (GPU accessible)
   * @param comp_buffer The location to output the compressed data to (GPU accessible).
   * @param decomp_buffer_size The length of the uncompressed input data
   * 
   */
  __global__ void bitcomp_header_k (CommonHeader *common_header, uint8_t* comp_buffer, uint64_t decomp_buffer_size)
  {
    common_header->magic_number = 0;
    common_header->major_version = NVCOMP_MAJOR_VERSION;
    common_header->minor_version = NVCOMP_MINOR_VERSION;
    common_header->format = FormatType::Bitcomp;
    common_header->decomp_data_size = decomp_buffer_size;
    common_header->num_chunks = 0;
    common_header->include_chunk_starts = false;
    common_header->full_comp_buffer_checksum = 0;
    common_header->decomp_buffer_checksum = 0;
    common_header->include_per_chunk_comp_buffer_checksums = false;
    common_header->include_per_chunk_decomp_buffer_checksums = false;
    common_header->uncomp_chunk_size = 0;
    common_header->comp_data_offset = (uintptr_t)comp_buffer - (uintptr_t)common_header;
  }

  /**
   * @brief Bitcomp compression helper 
   * 
   * @param common_header header filled in by this routine (GPU accessible)
   * @param decomp_buffer The uncompressed input data (GPU accessible)
   * @param decomp_buffer_size The length of the uncompressed input data
   * @param comp_buffer The location to output the compressed data to (GPU accessible).
   * @param comp_config Resulted from configure_compression given this decomp_buffer_size.
   * 
   * @param is_lossy if is lossy, 1: is, 0: not.
   * @param bitcomp_mode The lossy compression mode, 1: BITCOMP_LOSSY_FP_TO_SIGNED, 2: BITCOMP_LOSSY_FP_TO_UNSIGNED.
   * @param fp_type Read the data as floating point 0: 16, 1: 32, 2: 64.
   * @param delta Delta used for the integer quantization of the data.
   * The floating point values are divided by the delta provided during the compression, and converted
   * to integers. These integers are then compressed with a lossless encoder.
   * Values that would overflow during quantization (e.g. large input values and a very small delta),
   * as well as NaN, +Inf, -Inf will be handled correctly by the compression.
   * The integers can be either signed or unsigned.
   * 
   */
  void BitcompSingleStreamManager::do_compress(
      CommonHeader* common_header,
      const uint8_t* decomp_buffer, 
      uint8_t* comp_buffer,
      const CompressionConfig& comp_config)
  {
    bitcompHandle_t handle;
    if(is_lossy == 0){
      CHECK_EQ(
        bitcompCreatePlan(
            &handle,
            comp_config.uncompressed_buffer_size,
            bitcomp_data_type(format_spec->data_type),
            BITCOMP_LOSSLESS,
            static_cast<bitcompAlgorithm_t>(format_spec->algo)),
        BITCOMP_SUCCESS);

      CHECK_EQ(bitcompSetStream(handle, user_stream), BITCOMP_SUCCESS);

      CHECK_EQ(
          bitcompCompressLossless(handle, decomp_buffer, comp_buffer),
          BITCOMP_SUCCESS);
    }
    else if(is_lossy == 1){
      if(fp_type == 0){
        // CHECK_EQ(
        //   bitcompCreatePlan(
        //       &handle,
        //       comp_config.uncompressed_buffer_size,
        //       BITCOMP_FP16_DATA,
        //       static_cast<bitcompMode_t>(bitcomp_mode),
        //       static_cast<bitcompAlgorithm_t>(format_spec->algo)),
        //   BITCOMP_SUCCESS);
      
        // CHECK_EQ(bitcompSetStream(handle, user_stream), BITCOMP_SUCCESS);

        // CHECK_EQ(
        //   bitcompCompressLossy_fp16(handle, reinterpret_cast<const half*>(decomp_buffer), comp_buffer, static_cast<half>(delta)),
        //   BITCOMP_SUCCESS);
      }
      else if(fp_type == 1){
        CHECK_EQ(
          bitcompCreatePlan(
              &handle,
              comp_config.uncompressed_buffer_size,
              BITCOMP_FP32_DATA, // static_cast<bitcompMode_t>(bitcomp_mode),
              BITCOMP_LOSSY_FP_TO_SIGNED,
              static_cast<bitcompAlgorithm_t>(format_spec->algo)),
          BITCOMP_SUCCESS);

        CHECK_EQ(bitcompSetStream(handle, user_stream), BITCOMP_SUCCESS);

        CHECK_EQ(
          bitcompCompressLossy_fp32(handle, reinterpret_cast<const float*>(decomp_buffer), comp_buffer, static_cast<float>(delta)),
          BITCOMP_SUCCESS);
      }
      else if(fp_type == 2){
        // CHECK_EQ(
        //   bitcompCreatePlan(
        //       &handle,
        //       comp_config.uncompressed_buffer_size,
        //       BITCOMP_FP64_DATA,
        //       static_cast<bitcompMode_t>(bitcomp_mode),
        //       static_cast<bitcompAlgorithm_t>(format_spec->algo)),
        //   BITCOMP_SUCCESS);

        // CHECK_EQ(bitcompSetStream(handle, user_stream), BITCOMP_SUCCESS);

        // CHECK_EQ(
        //   bitcompCompressLossy_fp64(handle, reinterpret_cast<const double*>(decomp_buffer), comp_buffer, static_cast<double>(delta)),
        //   BITCOMP_SUCCESS);
      }
    }
    

    bitcomp_header_k<<<1, 1, 0, user_stream>>>(
        common_header, comp_buffer, comp_config.uncompressed_buffer_size);

    CHECK_EQ(
        bitcompGetCompressedSizeAsync(
            comp_buffer, &common_header->comp_data_size, user_stream),
        BITCOMP_SUCCESS);

    CHECK_EQ(bitcompDestroyPlan(handle), BITCOMP_SUCCESS);
  }

  /**
   * @brief Bitcomp decompression helper 
   *
   * @param decomp_buffer The location to output the decompressed data to (GPU accessible).
   * @param comp_buffer The compressed input data (GPU accessible).
   * @param decomp_config Resulted from configure_decompression given this decomp_buffer_size.
   */
  void BitcompSingleStreamManager::do_decompress(
      uint8_t* decomp_buffer, 
      const uint8_t* comp_buffer,
      const DecompressionConfig& config)
  {
    bitcompHandle_t handle;
    CHECK_EQ(
        bitcompCreatePlan(
            &handle,
            config.decomp_data_size,
            BITCOMP_FP32_DATA, // bitcomp_data_type(format_spec->data_type),
            BITCOMP_LOSSY_FP_TO_SIGNED, // BITCOMP_LOSSLESS,
            static_cast<bitcompAlgorithm_t>(format_spec->algo)),
        BITCOMP_SUCCESS);

    CHECK_EQ(bitcompSetStream(handle, user_stream), BITCOMP_SUCCESS);

    CHECK_EQ(bitcompUncompress(handle, comp_buffer, decomp_buffer), BITCOMP_SUCCESS);

    CHECK_EQ(bitcompDestroyPlan(handle), BITCOMP_SUCCESS);
  }

  /**
   * @brief Computes the maximum compressed output size for a given
   * uncompressed buffer.
   */
  size_t BitcompSingleStreamManager::calculate_max_compressed_output_size(CompressionConfig& comp_config)
  {
    return bitcompMaxBuflen (comp_config.uncompressed_buffer_size);
  }

} // namespace nvcomp

#else // ENABLE_BITCOMP

namespace nvcomp {
void BitcompSingleStreamManager::do_compress(CommonHeader*, const uint8_t*, uint8_t*, const CompressionConfig&)
{
  throw NVCompException(nvcompErrorNotSupported, "Bitcomp support not available in this build.");
}
void BitcompSingleStreamManager::do_decompress(uint8_t*, const uint8_t*, const DecompressionConfig&)
{
  throw NVCompException(nvcompErrorNotSupported, "Bitcomp support not available in this build.");
}
size_t BitcompSingleStreamManager::calculate_max_compressed_output_size(CompressionConfig&)
{
  throw NVCompException(nvcompErrorNotSupported, "Bitcomp support not available in this build.");
}
}

#endif // ENABLE_BITCOMP